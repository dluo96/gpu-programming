// Sum Reduction: 7 versions (implementations) 
// with different optimization strategies

#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Kernel v1: Interleaved Addressing with Divergent Branches.
// Disadvantages: thread divergence within warps are inefficient 
// and the modulo (%) operator is slow. 
__global__ void sum_reduction_v1(int *g_input, int *g_output, int numElements) {
    // Allocate dynamic shared memory
    extern __shared__ unsigned int sdata[];

    // Global (relative to grid) and local (relative to block) thread IDs
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ltid = threadIdx.x;

    // Each thread loads one element from global to shared memory
     if (tid < numElements) {
        sdata[ltid] = g_input[tid];
    } else {
        sdata[ltid] = 0;
    }
    __syncthreads();

    // Perform reduction in shared memory. Stride is doubled every iteration
    // (halving the number of threads used within the given block).
    // The interleaved addressing leads to large thread divergence because
    // threads are active/idle depending on whether their thread IDs are
    // powers of 2. The if-statement causes thread divergence within a warp. 
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        if(ltid % (2 * s) == 0) {
            sdata[ltid] += sdata[ltid + s];
        }
        __syncthreads();
    }

    // Write result for this block from shared to global memory
    if (ltid == 0) {
        g_output[blockIdx.x] = sdata[0];
    }
}

// Kernel version 2: Interleaved Addressing with Bank Conflicts.
// Compared to Version 1, this kernel replaces the divergent
// branch in the inner loop with a strided index and non-divergent 
// branch. This leads to a new drawback: shared memory bank conflicts. 
__global__ void sum_reduction_v2(int *g_input, int *g_output, int numElements) {
    extern __shared__ unsigned int sdata[];
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ltid = threadIdx.x;

     if (tid < numElements) {
        sdata[ltid] = g_input[tid];
    } else {
        sdata[ltid] = 0;
    }
    __syncthreads();

    // Perform reduction in shared memory. Still uses interleaved addressing,
    // but threads being active/idle no longer depends on whether thread IDs
    // are powers of 2. Consecutive thread IDs now run, solving the issue of
    // threads diverging within a warp. 
    // However, it introduces shared memory bank conflicts, which occur when
    // multiple threads in a given warp access different address locations
    // within the same bank. When this happens, the accesses serialize
    // rather than happening in parallel, thus reducing throughput.
    // Note: in NVIDIA GPUs, shared memory is divided into equally sized 
    // memory modules called banks. For many architectures, shared memory has
    // 32 banks, and each bank can service one memory request per clock cycle 
    // without conflicts.
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        // Index of element (in shared memory) to update
        unsigned int updateIdx = 2 * s * ltid;

        if(updateIdx < blockDim.x) {
            sdata[updateIdx] += sdata[updateIdx + s];
        }
        __syncthreads();
    }

    // Write result for this block from shared to global memory
    if (ltid == 0) {
        g_output[blockIdx.x] = sdata[0];
    }
}

// Kernel version 3: Sequential Addressing.
// Compared to Version 2, this kernel replaces the strided indexing
// in the inner loop with a reversed loop and thread-ID-based indexing.
// Advantages: the above means sequential addressing is conflict free.
// Disadvantages: half of the threads are idle on the 1st iteration, 
// three quarters of the threads are idle on the 2nd iteration, etc. 
__global__ void sum_reduction_v3(int *g_input, int *g_output, int numElements) {
    extern __shared__ unsigned int sdata[];
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ltid = threadIdx.x;

    if (tid < numElements) {
        sdata[ltid] = g_input[tid];
    } else {
        sdata[ltid] = 0;
    }
    __syncthreads();

    // Perform reduction in shared memory. 
    // Sequential addressing solves the shared memory bank conflicts
    // because the threads now access shared memory with a stride of
    // one 32-bit word (unsigned int) now. 
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(ltid < s) {
            sdata[ltid] += sdata[ltid + s];
        }
        __syncthreads();
    }

    // Write result for this block from shared to global memory
    if (ltid == 0) {
        g_output[blockIdx.x] = sdata[0];
    }
}

// Kernel version 4: First Sum During Load from Global Memory
__global__ void sum_reduction_v4(int *g_input, int *g_output, int numElements) {
    extern __shared__ unsigned int sdata[];

    // Halve the number of thread blocks: instead of a single load,
    // each thread loads 2 elements from global memory, sums them, and
    // loads the result into shared memory.
    unsigned int tid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int ltid = threadIdx.x;
    if (tid < numElements) {
        // Do first sum during load
        sdata[ltid] = g_input[tid] + g_input[tid + blockDim.x];
    } else {
        sdata[ltid] = 0;
    }
    __syncthreads();

    // Same as version 3
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(ltid < s) {
            sdata[ltid] += sdata[ltid + s];
        }
        __syncthreads();
    }

    // Write result for this block from shared to global memory
    if (ltid == 0) {
        g_output[blockIdx.x] = sdata[0];
    }
}

// Kernel version 5
// TODO: fix with sum_reduction_v5.cu
__global__ void sum_reduction_v5(int *g_input, int *g_output, int numElements) {
    extern __shared__ unsigned int sdata[];
    unsigned int tid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int ltid = threadIdx.x;
    if (tid < numElements) {
        sdata[ltid] = g_input[tid] + g_input[tid + blockDim.x];
    } else {
        sdata[ltid] = 0;
    }
    __syncthreads();

    for(unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if(ltid < s) {
            sdata[ltid] += sdata[ltid + s];
        }
        __syncthreads();
    }

    // Warp reduce
    if(ltid < 32) {
    	sdata[ltid] += sdata[ltid + 32];
		sdata[ltid] += sdata[ltid + 16];
		sdata[ltid] += sdata[ltid + 8];
		sdata[ltid] += sdata[ltid + 4];
		sdata[ltid] += sdata[ltid + 2];
		sdata[ltid] += sdata[ltid + 1];
    }

    // Write result for this block from shared to global memory
    if (ltid == 0) {
        g_output[blockIdx.x] = sdata[0];
    }
}

void init_vector(int *a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = 1;  // Initializing with 1 for simplicity
    }
}

int main() {
    int N = 1 << 24;
    size_t bytes = N * sizeof(int);

    int *input, *result;
    int *d_input, *d_result;
    input = (int*)malloc(bytes);
    result = (int*)malloc(bytes);
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_result, bytes);
    init_vector(input, N);
    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

    // Threads, blocks, grids, and dynamic shared memory
    int threadsPerBlock = 128;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    int sizeSharedMemory = threadsPerBlock * sizeof(int);

    // CUDA events for timing kernels
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // Kernel decomposition with recursion
    hipEventRecord(start);
    sum_reduction_v4<<<blocksPerGrid/2, threadsPerBlock, sizeSharedMemory>>>(d_input, d_result, N);
    hipDeviceSynchronize();
    unsigned int numPartialSums = blocksPerGrid;
    while(numPartialSums > 1) {
        int nBlocks = (numPartialSums + threadsPerBlock - 1) / threadsPerBlock;
        // printf("Partial sums computed = %i. Threads per block = %i. Blocks required = %i.\n", 
        //         numPartialSums, threadsPerBlock, nBlocks);
        sum_reduction_v4<<<nBlocks/2, threadsPerBlock, sizeSharedMemory>>>(d_result, d_result, numPartialSums);
        hipDeviceSynchronize();
        numPartialSums = nBlocks;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    assert(result[0] == N);
    printf("Success! Computed sum reduction.\n");
    printf("Result: %d\n", result[0]);
    printf("Time elapsed: %f milliseconds\n", milliseconds);

    // Cleanup
    hipFree(d_input);
    hipFree(d_result);
    free(input);
    free(result);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}